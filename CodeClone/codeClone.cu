#include "hip/hip_runtime.h"
#include <iterator>
#include <vector>
#include <sstream> 
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string>
#include <stdlib.h>


//Cuda error checking - non mandatory
void cudaCheckError() {
 hipError_t e=hipGetLastError();
 if(e!=hipSuccess) {
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));
   exit(0); 
 }
}

//GPU kernel
__global__
void doDetection_cudaVersion(int **vector_array, int N,  int* result_clones){
               *result_clones=210;   

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int e = blockIdx.x * blockDim.x + threadIdx.x; 
    if(e > i){
        if(vector_array[i][0] == vector_array[e][0] &&
               vector_array[i][1] == vector_array[e][1] &&
               vector_array[i][2] == vector_array[e][2] &&
               vector_array[i][3] == vector_array[e][3] &&
               vector_array[i][4] == vector_array[e][4] &&
               vector_array[i][5] == vector_array[e][5] &&
               vector_array[i][6] == vector_array[e][6]){
               (*result_clones)++;   
         }
    }
}


std::vector<std::vector<int>> doDetection(std::vector<std::vector<int>> vector_array){

    int i, j;
    std::vector<std::vector<int>> result_array;

       
    for(i = 0; i < (int) vector_array.size(); ++i){

        result_array.push_back(std::vector<int>());
    }

    for(i = 0; i < (int) vector_array.size(); ++i){

        for(j = i+1; j < (int) vector_array.size(); ++j){

            if(vector_array[i][0] == vector_array[j][0] &&
               vector_array[i][1] == vector_array[j][1] &&
               vector_array[i][2] == vector_array[j][2] &&
               vector_array[i][3] == vector_array[j][3] &&
               vector_array[i][4] == vector_array[j][4] &&
               vector_array[i][5] == vector_array[j][5] &&
               vector_array[i][6] == vector_array[j][6]){


                result_array[i].push_back(j);

            }
        }
    }

    return result_array;
}



int main(int argc,char **argv){
    //initialize data
    std::vector<std::vector<int>> vector_array;
    std::string line;
    std::ifstream myfile ("./metricas.txt");
    getline(myfile, line);
    myfile.close();

    std::vector<int> vect;
    std::stringstream ss(line);

    int limitM;

    while (ss >> limitM)
    {
        vect.push_back(limitM);

        if (ss.peek() == ',')
            ss.ignore();
    }

    vector_array.resize(vect.size()/7);
    for(int vecI=0; vecI < (vect.size()/7); vecI++){
        vector_array[vecI].resize(7);
    }
    int vector_array_h[vect.size()/7][7];
    for(int x=0; x< (vect.size()/7);x++){
        for(int y=0; y<7; y++){
                vector_array[x][y]=vect[x*7+y];
                vector_array_h[x][y]=vect[x*7+y];
        }
    }

    printf("Begin \n");
    //Block size and number
    int block_size, block_no;

    //memory allocation
    int** vector_array_d;

    block_size = 100; //threads per block
    block_no = vector_array.size()/block_size;
    
    //Work definition
    dim3 dimBlock(vector_array.size(), vector_array.size(),1);
    dim3 dimGrid(1,1,1);


    int *numClones = (int*)malloc(sizeof(int));;
    *numClones=10;
    int numClones_d =0;
    int *numClones_dptr;

    printf("Allocating device memory on host..\n");
    //GPU memory allocation
    hipMalloc((void **) &vector_array_d, vector_array.size()*7*sizeof(int));
    hipMalloc(&numClones_dptr, sizeof(int));
    memset(numClones,0,sizeof(int));

    printf("Copying to device..\n");
    hipMemcpy(vector_array_d, vector_array_h, vector_array.size()*7*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(numClones_dptr, numClones, sizeof(int),hipMemcpyHostToDevice);

    clock_t start_d=clock();
    printf("Doing GPU clone detection... \n");
    doDetection_cudaVersion<<<dimBlock, dimGrid>>>(vector_array_d,vector_array.size(), numClones_dptr);
    cudaCheckError();

    //Wait for kernel call to finish
    hipDeviceSynchronize();

    clock_t end_d = clock();
    

    printf("Doing CPU clone detection... \n");
    clock_t start_h = clock();
    std::vector<std::vector<int>> result_array  = doDetection(vector_array);
    clock_t end_h = clock();
    std::cout << "Cantidad de clones encontrados serial: " << result_array[0].size() << "\n";


	
    //Time computing
    double time_d = (double)(end_d-start_d)/CLOCKS_PER_SEC;
    double time_h = (double)(end_h-start_h)/CLOCKS_PER_SEC;

    //Copying data back to host, this is a blocking call and will not start until all kernels are finished
    hipMemcpy(numClones, numClones_dptr, sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "Cantidad de clones encontrados paralelo: " << *numClones << "\n";
    printf("GPU time = %fs \t CPU time = %fs\n", time_d, time_h);

    //Free GPU memory
    free(numClones);
    hipFree(numClones_dptr);
    hipFree(vector_array_d);
    return 0;
}
